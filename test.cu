#include "hip/hip_runtime.h"
#include <cstring>
#include <limits>
#include <stdexcept>
#include <string>
#include <algorithm>

class string_view final {
 public:
  static constexpr int npos = int(-1);

  constexpr int find() const noexcept {
    substr_(1).equals_(substr_(1));
    return 0;
  }
  constexpr string_view substr_(int pos = 0, int count = npos)
      const {
    return string_view{begin_ + pos, std::min(count, size_ - pos)};
  }

  constexpr bool equals_(string_view rhs) const {
    return size_ == rhs.size_ && 0 == __builtin_memcmp(begin_, rhs.begin_, size_);
  }

  const char *begin_ = "12345";
  int size_ = 5;
};

int main() {
  constexpr auto x = string_view().find();
}
