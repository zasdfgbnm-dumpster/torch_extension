#include "hip/hip_runtime.h"
#include <cstring>
#include <limits>
#include <stdexcept>
#include <string>
#include <algorithm>

template <class CharT>
class basic_string_view final {
 public:
  static constexpr int npos = int(-1);

  constexpr basic_string_view() noexcept : begin_(nullptr), size_(0) {}

  explicit constexpr basic_string_view(const char *str, int count)
      : begin_(str), size_(count) {}

  /* implicit */ constexpr basic_string_view(const char *str)
      : basic_string_view(str, strlen_(str)) {}

  constexpr int find() const noexcept {
    substr_(1).equals_(substr_(1));
    return 0;
  }

  constexpr int find(const char *s, int pos = 0) const {
    return find(basic_string_view(s), pos);
  }

  static constexpr int strlen_(const char *str) noexcept {
    const char *current = str;
    while (*current != '\0') {
      ++current;
    }
    return current - str;
  }

  constexpr basic_string_view substr_(int pos = 0, int count = npos)
      const {
    return basic_string_view{begin_ + pos, std::min(count, size_ - pos)};
  }

  constexpr bool equals_(basic_string_view rhs) const {
    return size_ == rhs.size_ && 0 == __builtin_memcmp(begin_, rhs.begin_, size_);
  }

  const char *begin_;
  int size_;
};

using string_view = basic_string_view<char>;

int main() {
  constexpr auto x = string_view("fsdfa::fsdfas").find();
}
