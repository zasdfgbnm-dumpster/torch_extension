
#include <hip/hip_runtime.h>
#include <cstring>
#include <limits>
#include <stdexcept>
#include <string>
#include <algorithm>

template <class CharT>
class basic_string_view final {
 public:
  using value_type = CharT;
  using pointer = CharT*;
  using const_pointer = const CharT*;
  using reference = CharT&;
  using const_reference = const CharT&;
  using const_iterator = const CharT*;
  using iterator = const_iterator;
  using size_type = std::size_t;
  using difference_type = std::ptrdiff_t;

  static constexpr size_type npos = size_type(-1);

  constexpr basic_string_view() noexcept : begin_(nullptr), size_(0) {}

  explicit constexpr basic_string_view(const_pointer str, size_type count)
      : begin_(str), size_(count) {}

  /* implicit */ constexpr basic_string_view(const_pointer str)
      : basic_string_view(str, strlen_(str)) {}

  constexpr size_type find() const noexcept {
    substr_(1).equals_(substr_(1));
    return 0;
  }

  constexpr size_type find(const_pointer s, size_type pos = 0) const {
    return find(basic_string_view(s), pos);
  }

  static constexpr size_type strlen_(const_pointer str) noexcept {
    const_pointer current = str;
    while (*current != '\0') {
      ++current;
    }
    return current - str;
  }

  constexpr basic_string_view substr_(size_type pos = 0, size_type count = npos)
      const {
    return basic_string_view{begin_ + pos, std::min(count, size_ - pos)};
  }

  constexpr bool equals_(basic_string_view rhs) const {
    return size_ == rhs.size_ && 0 == __builtin_memcmp(begin_, rhs.begin_, size_);
  }

  const_pointer begin_;
  size_type size_;
};

using string_view = basic_string_view<char>;

int main() {
  constexpr auto x = string_view("fsdfa::fsdfas").find();
}
